#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include<sys/time.h>
#include <hip/hip_runtime.h>

#define XSIZE 2560
#define YSIZE 2048
#define BLOCKY 32
#define BLOCKX 32
#define MAXITER 255

double xleft = -2.01, xright = 1, yupper, ylower, ycenter = 1e-6, step;
int host_pixel[XSIZE * YSIZE];
int device_pixel[XSIZE * YSIZE];

#define PIXEL(i,j) ((i) + (j) * XSIZE)

typedef struct {
	double real, imag;
} complex_t;

typedef unsigned char uchar;

// ********** SUBTASK1: Create kernel device_calculate ******************/
//Insert code here
// Hint: Use _global_ for the kernal function to be executed on the GPU.
// Also set up a single grid with a 2D thread block
__global__ void device_calculate(int* device_pixel, double xleft, double step, double yupper) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if(i < XSIZE && j < YSIZE) {
		complex_t c, z, temp;
		int iter = 0;
		c.real = (xleft + step * i);
		c.imag = (yupper - step * j);
		z = c;
		while (z.real * z.real + z.imag * z.imag < 4.0) {
			temp.real = z.real * z.real - z.imag * z.imag + c.real;
			temp.imag = 2.0 * z.real * z.imag + c.imag;
			z = temp;
			if (++iter == MAXITER) break;
		}
		device_pixel[PIXEL(i, j)] = iter;
	}
}
// ********** SUBTASK1 END ***********************************************/

void host_calculate() {

	for(int j = 0; j < YSIZE; j++) {
		for(int i = 0; i < XSIZE; i++) {
			// Calculate the number of iterations until divergence for each pixel.
			// If divergence never happens, return MAXITER
			complex_t c, z, temp;
			int iter = 0;
			c.real = (xleft + step * i);
			c.imag = (yupper - step * j);
			z = c;
			while (z.real * z.real + z.imag * z.imag < 4.0) {
				temp.real = z.real * z.real - z.imag * z.imag + c.real;
				temp.imag = 2.0 * z.real * z.imag + c.imag;
				z = temp;
				if (++iter == MAXITER) break;
			}
			host_pixel[PIXEL(i, j)] = iter;
		}
	}

}

// save 24-bits bmp file, buffer must be in bmp format: upside-down
void savebmp(char* name, uchar* buffer, int x, int y) {

	FILE* f = fopen(name, "wb");
	if(!f) {
		printf("Error writing image to disk.\n");
		return;
	}
	unsigned int size = x * y * 3 + 54;
	uchar header[54]={'B', 'M', size & 255, (size >> 8) & 255, (size >> 16) & 255, size >> 24, 0, 0, 0, 0, 54, 0, 0, 0, 40, 0, 0, 0, x & 255, x >> 8, 0, 0, y & 255, y >> 8, 0, 0, 1, 0, 24, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
	fwrite(header, 1, 54, f);
	fwrite(buffer, 1, XSIZE * YSIZE * 3, f);
	fclose(f);

}

// given iteration number, set a colour
void fancycolour(uchar* p, int iter) {

    if(iter == MAXITER);
    else if(iter < 8) {
        p[0] = 128 + iter * 16;
        p[1] = p[2] = 0;
    } else if(iter < 24) {
        p[0] = 255;
        p[1] = p[2] = (iter - 8) * 16;
    } else if(iter < 160) {
        p[0] = p[1] = 255 - (iter - 24) * 2;
        p[2] = 255;
    } else {
        p[0] = p[1] = (iter - 160) * 2;
        p[2] = 255 - (iter - 160) * 2;
    }

}

// Get system time to microsecond precision (similar to MPI_Wtime), returns time in seconds
double walltime(void) {

	static struct timeval t;
	gettimeofday(&t, NULL);
	return (t.tv_sec + 1e-6 * t.tv_usec);

}

int main(int argc, char** argv) {

	if(argc == 1) {
		puts("Usage: MANDEL n");
		puts("n decides whether image should be written to disk (1=yes, 0=no)");
		return 0;
	}

	double start, hosttime = 0, devicetime = 0, memtime = 0;

	hipDeviceProp_t p;
	hipSetDevice(0);
	hipGetDeviceProperties(&p, 0);
	printf("Device compute capability: %d.%d\n", p.major, p.minor);

	/* Calculate the range in the y-axis such that we preserve the aspect ratio */
	step = (xright - xleft) / XSIZE;
	yupper = ycenter + (step * YSIZE) / 2;
	ylower = ycenter - (step * YSIZE) / 2;

	/* Host calculates image */
	start = walltime();
	host_calculate();
	hosttime += walltime() - start;

	//********** SUBTASK2: Set up device memory ***************************/
	// Insert code here
	int* d_device_pixel;
	hipMalloc((void**)&d_device_pixel, XSIZE * YSIZE * sizeof(int));
	/********** SUBTASK2 END **********************************************/

	start = walltime();

	//********* SUBTASK3: Execute the kernel on the device ************/
	// Insert code here
	dim3 block(BLOCKX, BLOCKY);
	dim3 grid((XSIZE + BLOCKX - 1) / BLOCKX, (YSIZE + BLOCKY - 1) / BLOCKY);
	device_calculate<<<grid, block>>>(d_device_pixel, xleft, step, yupper);
	hipDeviceSynchronize();
	//********** SUBTASK3 END *****************************************/

	devicetime += walltime() - start;
	start = walltime();

	//***** SUBTASK4: Transfer the result from device to device_pixel[][]*/
	// Insert code here
	hipMemcpy(device_pixel, d_device_pixel, XSIZE * YSIZE * sizeof(int), hipMemcpyDeviceToHost);
	//********** SUBTASK4 END ******************************************/

	memtime += walltime() - start;

	/****** SUBTASK5: Free the device memory also ************************/
	// Insert code here
	hipFree(d_device_pixel);
	/********** SUBTASK5 END ******************************************/

	int errors = 0;
	// check if result is correct
	for(int i = 0; i < XSIZE; i++) {
		for(int j = 0; j < YSIZE; j++) {
			int diff = host_pixel[PIXEL(i, j)] - device_pixel[PIXEL(i, j)];
			if(diff < 0) diff = -diff;
			// allow +-1 difference
			if(diff > 1) {
				if(errors < 10) 
					printf("Error on pixel %d %d: expected %d, found %d\n", i, j, host_pixel[PIXEL(i, j)], device_pixel[PIXEL(i, j)]);
				else if(errors == 10) 
					puts("...");
				errors++;
			}
		}
	}

	if(errors > 0) 
		printf("Found %d errors.\n", errors);
	else 
		puts("Device calculations are correct.");

	printf("\n");
	printf("Host time: %7.3f ms\n", hosttime * 1e3);
	printf("Device calculation: %7.3f ms\n", devicetime * 1e3);
	printf("Copy result: %7.3f ms\n", memtime * 1e3);

	if(strtol(argv[1], NULL, 10) != 0) {
		// create nice image from iteration counts. take care to create it upside down (bmp format)
		unsigned char *buffer = (unsigned char *)calloc(XSIZE * YSIZE * 3, 1);
		for(int i = 0; i < XSIZE; i++) {
			for(int j = 0; j < YSIZE; j++) {
				int p = ((YSIZE - j - 1) * XSIZE + i) * 3;
				fancycolour(buffer + p, device_pixel[PIXEL(i, j)]);
			}
		}
		// write image to disk
		savebmp("mandel1.bmp", buffer, XSIZE, YSIZE);
	}

	return 0;

}